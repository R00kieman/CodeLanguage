#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#ifdef _WIN32
#include <windows.h>
#else
#include <sys/time.h>
#endif

#include "cudastart.h"


double cpuSecond()
{
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return((double)tp.tv_sec+(double)tp.tv_usec*1e-6);

}

void initialData(float* ip,int size)
{
    time_t t;
    srand((unsigned )time(&t));
    for(int i=0;i<size;i++)
    {
        ip[i]=(float)(rand()&0xffff)/1000.0f;
    }
}

void initDevice(int devNum)
{
    int dev = devNum;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp,dev));
    printf("Using device %d: %s\n",dev,deviceProp.name);
    printf("warpSize: %d\n", deviceProp.warpSize);
    CHECK(hipSetDevice(dev));

}

void checkResult(float * hostRef,float * gpuRef,const int N)
{
    double epsilon=1.0E-8;
    for(int i=0;i<N;i++)
    {
        if(abs(hostRef[i]-gpuRef[i])>epsilon)
        {
            printf("Results don\'t match!\n");
            printf("%f(hostRef[%d] )!= %f(gpuRef[%d])\n",hostRef[i],i,gpuRef[i],i);
            return;
        }
    }
    printf("Check result success!\n");
}

//CPU对照组，用于对比加速比
void sumMatrix2DonCPU(float * MatA,float * MatB,float * MatC,int nx,int ny)
{
    float* a = MatA;
    float* b = MatB;
    float* c = MatC;
    for(int j=0; j<ny; j++)
    {
        for(int i=0; i<nx; i++)
        {
            c[i] = a[i]+b[i];
        }
        c += nx;
        b += nx;
        a += nx;
    }
}

//核函数，每一个线程计算矩阵中的一个元素。
__global__ void sumMatrix(float * MatA,float * MatB,float * MatC,int nx,int ny)
{
    /*
     * gridDim: 这是一个三维变量，用于表示一个网格的维度。CUDA中的线程结构可以被组织成一个三维的网格，每个维度可以有多个块（blocks）。
     *          gridDim.x 表示网格在 x 维度上的块数，gridDim.y 表示网格在 y 维度上的块数，gridDim.z 表示网格在 z 维度上的块数。
     * blockDim: 这也是一个三维变量，用于表示一个块的维度。每个块都可以包含多个线程。
     *          blockDim.x 表示块在 x 维度上的线程数，blockDim.y 表示块在 y 维度上的线程数，blockDim.z 表示块在 z 维度上的线程数。
     * blockIdx: 这是一个三维变量，用于表示当前线程所属的块（block）在整个网格（grid）中的索引。
     *          blockIdx.x 表示当前线程所属的块在 x 维度上的索引，blockIdx.y 表示在 y 维度上的索引，blockIdx.z 表示在 z 维度上的索引。
     * threadIdx: 这也是一个三维变量，用于表示当前线程在其所属的块中的索引。
     *          每个块都可以包含多个线程，而 threadIdx.x 表示当前线程在 x 维度上的索引，threadIdx.y 表示在 y 维度上的索引，threadIdx.z 表示在 z 维度上的索引。
     * */
    int ix = threadIdx.x+blockDim.x*blockIdx.x;
    int iy = threadIdx.y+blockDim.y*blockIdx.y;
    int idx = ix+iy*ny;
    if (ix<nx && iy<ny)
    {
        MatC[idx] = MatA[idx]+MatB[idx];
    }
}

//主函数
int main(int argc,char** argv)
{
    //设备初始化
    printf("strating...\n");
    initDevice(0);

    //输入二维矩阵，4096*4096，单精度浮点型。
    int nx = 1<<12;
    int ny = 1<<12;
    int nBytes = nx*ny*sizeof(float);

    //Malloc，开辟主机内存
    float* A_host = (float*)malloc(nBytes);
    float* B_host = (float*)malloc(nBytes);
    float* C_host = (float*)malloc(nBytes);
    float* C_from_gpu = (float*)malloc(nBytes);
    initialData(A_host, nx*ny);
    initialData(B_host, nx*ny);

    //hipMalloc，开辟设备内存
    float* A_dev = NULL;
    float* B_dev = NULL;
    float* C_dev = NULL;
    CHECK(hipMalloc((void**)&A_dev, nBytes));
    CHECK(hipMalloc((void**)&B_dev, nBytes));
    CHECK(hipMalloc((void**)&C_dev, nBytes));

    //输入数据从主机内存拷贝到设备内存
    CHECK(hipMemcpy(A_dev, A_host, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(B_dev, B_host, nBytes, hipMemcpyHostToDevice));

    //二维线程块，32×32
    dim3 block(32, 32);
    //二维线程网格，128×128
    dim3 grid((nx-1)/block.x+1, (ny-1)/block.y+1);

    //测试GPU执行时间
    double gpuStart = cpuSecond();
    printf("before sumMatrix\n");
    //将核函数放在线程网格中执行 (在调用内核函数的时候，会在<<< >>>内设置两个参数，分别代表线程网格的维度和线程块的维度。)
    sumMatrix<<<grid,block>>>(A_dev, B_dev, C_dev, nx, ny);
    printf("after sumMatrix\n");
    CHECK(hipDeviceSynchronize());
    double gpuTime = cpuSecond() - gpuStart;
    printf("GPU Execution Time: %f sec\n", gpuTime);

    //在CPU上完成相同的任务
    hipMemcpy(C_from_gpu, C_dev, nBytes, hipMemcpyDeviceToHost);
    double cpuStart=cpuSecond();
    sumMatrix2DonCPU(A_host, B_host, C_host, nx, ny);
    double cpuTime = cpuSecond() - cpuStart;
    printf("CPU Execution Time: %f sec\n", cpuTime);

    //检查GPU与CPU计算结果是否相同
    CHECK(hipMemcpy(C_from_gpu, C_dev, nBytes, hipMemcpyDeviceToHost));
    checkResult(C_host, C_from_gpu, nx*ny);

    //释放内存
    hipFree(A_dev);
    hipFree(B_dev);
    hipFree(C_dev);
    free(A_host);
    free(B_host);
    free(C_host);
    free(C_from_gpu);
    hipDeviceReset();
    return 0;
}